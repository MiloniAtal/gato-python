/*
 *
 *  GATO linear system solver
 *
 */

#include <iostream>
#include <stdio.h>
#include <assert.h>

#include "include/gato_defines.h"
#include "include/types.h"
#include "src/gato_utils.cuh"
#include "src/gato_schur.cuh"
#include "src/gato_pcg.cuh"

#include <sstream>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#include <hip/hip_runtime.h>

namespace py = pybind11;
int gato_linsys(int *d_G_row, int *d_G_col, float *d_G_val,
                int *d_C_row, int *d_C_col, float *d_C_val,
                float *d_g_val,
                float *d_c_val,
                float *lambda, float *dz,
                bool warm_start, float eps, int max_iter){

    float *d_S, *d_Pinv, *d_gamma, *d_lambda, *d_dz;
    int pcg_iters;
    float rho = .001;

    float *d_G_dense, *d_C_dense;
    cuda_calloc((void **)&d_G_dense, KKT_G_DENSE_SIZE_BYTES);
    cuda_calloc((void **)&d_C_dense, KKT_C_DENSE_SIZE_BYTES);


    cuda_malloc((void **)&d_S,     3*STATES_SQ*KNOT_POINTS*sizeof(float));
    cuda_malloc((void **)&d_Pinv,  3*STATES_SQ*KNOT_POINTS*sizeof(float));
    cuda_malloc((void **)&d_gamma, STATE_SIZE*KNOT_POINTS*sizeof(float));
    cuda_malloc((void **)&d_dz, ((STATES_S_CONTROLS)*KNOT_POINTS-CONTROL_SIZE)*sizeof(float));

    cuda_calloc((void **)&d_lambda, STATE_SIZE*KNOT_POINTS*sizeof(float));
    if(warm_start)
        gpuErrchk( hipMemcpy(d_lambda, lambda, STATE_SIZE*KNOT_POINTS*sizeof(float), hipMemcpyHostToDevice));

#if DEBUG_MODE
        hipDeviceSynchronize();
#endif

    form_schur(d_G_row, d_G_col, d_G_val, d_G_dense,
               d_C_row, d_C_col, d_C_val, d_C_dense,
               d_g_val,
               d_c_val,
               d_S, d_Pinv, d_gamma, rho);
    
    pcg_iters = solve_pcg<float>(d_S, d_Pinv, d_gamma, d_lambda, warm_start, eps, max_iter);

#if DEBUG_MODE
        hipDeviceSynchronize();
#endif

    compute_dz(d_G_dense, d_C_dense, d_g_val, d_lambda, d_dz);

#if DEBUG_MODE
    hipDeviceSynchronize();
#endif

    gpuErrchk(hipMemcpy(lambda, d_lambda, STATE_SIZE*KNOT_POINTS*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(dz, d_dz, (STATES_S_CONTROLS*KNOT_POINTS-CONTROL_SIZE)*sizeof(float), hipMemcpyDeviceToHost));

    cuda_free((void **)d_G_dense);
    cuda_free((void **)d_C_dense);

    gpuErrchk(hipFree(d_S));
    gpuErrchk(hipFree(d_Pinv));
    gpuErrchk(hipFree(d_gamma));
    gpuErrchk(hipFree(d_lambda));
    gpuErrchk(hipFree(d_dz));
    return pcg_iters;
}

py::tuple main_call(std::vector<int> sG_indptr_vector, std::vector<int> sG_indices_vector, std::vector<float> sG_data_vector, 
                std::vector<int> sC_indptr_vector, std::vector<int> sC_indices_vector, std::vector<float> sC_data_vector, 
                std::vector<float> g_vector, std::vector<float> c_vector, std::vector<float> input_lambda, int testiters, float exit_tol, int max_iters, bool warm_start){
    
    
    int* G_row = sG_indptr_vector.data();
    int G_row_size_bytes = sG_indptr_vector.size() * sizeof(int);
    
    int* G_col = sG_indices_vector.data();
    int G_col_size_bytes = sG_indices_vector.size() * sizeof(int);

    float* G_val = sG_data_vector.data();
    int G_val_size_bytes = sG_data_vector.size() * sizeof(float);
    
    int* C_row = sC_indptr_vector.data();
    int C_row_size_bytes = sC_indptr_vector.size() * sizeof(int);
    
    int* C_col = sC_indices_vector.data();
    int C_col_size_bytes = sC_indices_vector.size() * sizeof(int);

    float* C_val = sC_data_vector.data();
    int C_val_size_bytes = sC_data_vector.size() * sizeof(float);
    
    float* g_val = g_vector.data();
    int g_size_bytes = g_vector.size() * sizeof(float);
    
    float* c_val = c_vector.data();
    int c_size_bytes = c_vector.size() * sizeof(float);

#if DEBUG_MODE
/*
    std::cout << "G row \n";
    for(unsigned i = 0; i < G_row_size_bytes/sizeof(int); i++){
        std::cout << G_row[i] << " ";
    }
    std::cout << "\nG col \n";
    for(unsigned i = 0; i < G_col_size_bytes/sizeof(int); i++){
        std::cout << G_col[i] << " ";
    }
    std::cout << "\nG val \n";
    for(unsigned i = 0; i < G_val_size_bytes/sizeof(int); i++){
        std::cout << G_val[i] << " ";
    }
*/
#endif  /* #if DEBUG_MODE */

    float dz[(STATES_S_CONTROLS)*KNOT_POINTS-CONTROL_SIZE];


    float *d_G_val, *d_C_val, *d_g_val, *d_c_val;
    int *d_G_row, *d_G_col, *d_C_row, *d_C_col;

    cuda_malloc((void **)&d_G_val, G_val_size_bytes);
    cuda_malloc((void **)&d_G_row, G_row_size_bytes);
    cuda_malloc((void **)&d_G_col, G_col_size_bytes);
    cuda_malloc((void **)&d_C_val, C_val_size_bytes);
    cuda_malloc((void **)&d_C_row, C_row_size_bytes);
    cuda_malloc((void **)&d_C_col, C_col_size_bytes);
    cuda_malloc((void **)&d_g_val, g_size_bytes);
    cuda_malloc((void **)&d_c_val, c_size_bytes);
    
    /// TODO: wrap cudamemcpy like malloc
    gpuErrchk( hipMemcpy(d_G_val, G_val, G_val_size_bytes, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_G_row, G_row, G_row_size_bytes, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_G_col, G_col, G_col_size_bytes, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_C_val, C_val, C_val_size_bytes, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_C_row, C_row, C_row_size_bytes, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_C_col, C_col, C_col_size_bytes, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_g_val, g_val, g_size_bytes, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_c_val, c_val, c_size_bytes, hipMemcpyHostToDevice));



    float lambda[STATE_SIZE*KNOT_POINTS];
    for(int i=0;i<STATE_SIZE*KNOT_POINTS;i++){
        lambda[i] = input_lambda[i];
    }

    float times[testiters];
    hipEvent_t start, stop;    

    for(int i = 0; i < testiters; i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        int iters = gato_linsys(d_G_row, d_G_col, d_G_val,
                                d_C_row, d_C_col, d_C_val,
                                d_g_val,
                                d_c_val,
                                lambda, dz,
                                warm_start, exit_tol, max_iters);

        hipEventRecord(stop);
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        
        float e1;
        hipEventElapsedTime(&e1, start, stop);
        times[i] = e1;

        if(i==0){
            printf("first run PCG terminated in %d iterations, time:  %f\n", iters, e1);
        }
    }

    for(int i = 1; i < testiters; i++){
        times[0] += times[i];
    }

    printf("avg time: %f\n", times[0]/testiters);

    
    // for(int i =0; i < STATE_SIZE*KNOT_POINTS; i++)
    //      printf("%f\n", lambda[i]);
    // printf("\n\ndz\n");
    // for(int i =0; i < (STATES_S_CONTROLS)*KNOT_POINTS-CONTROL_SIZE; i++)
    //     printf("%f\n", dz[i]);

    hipEventDestroy(start);
    hipEventDestroy(stop);


    gpuErrchk( hipFree(d_G_val));
    gpuErrchk( hipFree(d_G_row));
    gpuErrchk( hipFree(d_G_col));
    gpuErrchk( hipFree(d_C_val));
    gpuErrchk( hipFree(d_C_row));
    gpuErrchk( hipFree(d_C_col));
    gpuErrchk( hipFree(d_g_val));
    gpuErrchk( hipFree(d_c_val));
    
    
	py::list p_lambda;
    for(int i=0;i<=STATE_SIZE*KNOT_POINTS;i++){
        p_lambda.append(lambda[i]);
    }
    py::list p_dz;
    for(int i=0;i<=(STATES_S_CONTROLS)*KNOT_POINTS-CONTROL_SIZE;i++){
        p_dz.append(dz[i]);
    }

    py::tuple ans = py::make_tuple(p_lambda, p_dz);
    return ans;

    
}

PYBIND11_MODULE(gpu_library, m)
{
  m.def("linsys_solve", &main_call, py::return_value_policy::move);
}
